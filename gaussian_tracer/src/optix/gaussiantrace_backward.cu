#include "hip/hip_runtime.h"
#include "gaussiantrace_backward.h"
#include <optix.h>


namespace raytracing {

extern "C" {
	__constant__ Gaussiantrace_backward::Params params;
}

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	float O_final = params.alpha[idx.x];
	if (O_final==0.0f) return;

	glm::vec3 ray_o = params.ray_origins[idx.x];
	glm::vec3 ray_d = params.ray_directions[idx.x];
	glm::vec3 ray_origin;
	glm::vec3 C = glm::vec3(0.0f, 0.0f, 0.0f), C_final = params.colors[idx.x], grad_colors = params.grad_colors[idx.x];
	float D = 0.0f, D_final = params.depths[idx.x], grad_depths = params.grad_depths[idx.x];
	float O = 0.0f, grad_alpha = params.grad_alpha[idx.x];

	float T = 1.0f, t_start = 0.0f, t_curr = 0.0f;

	HitInfo hitArray[MAX_BUFFER_SIZE];
	unsigned int hitArrayPtr0 = (unsigned int)((uintptr_t)(&hitArray) & 0xFFFFFFFF);
    unsigned int hitArrayPtr1 = (unsigned int)(((uintptr_t)(&hitArray) >> 32) & 0xFFFFFFFF);

	int k=0;
	while ((t_start < T_SCENE_MAX) && (T > params.transmittance_min)){
		k++;
		ray_origin = ray_o + t_start * ray_d;
		
		for (int i = 0; i < MAX_BUFFER_SIZE; ++i) {
			hitArray[i].t = 1e16f;
			hitArray[i].primIdx = -1;
		}
		optixTrace(
			params.handle,
			make_float3(ray_origin.x, ray_origin.y, ray_origin.z),
			make_float3(ray_d.x, ray_d.y, ray_d.z),
			0.0f,                // Min intersection distance
			T_SCENE_MAX,               // Max intersection distance
			0.0f,                // rayTime -- used for motion blur
			OptixVisibilityMask(255), // Specify always visible
			OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
			0,                   // SBT offset
			1,                   // SBT stride
			0,                   // missSBTIndex
			hitArrayPtr0,
			hitArrayPtr1
		);

		for (int i = 0; i < MAX_BUFFER_SIZE; ++i) {
			int primIdx = hitArray[i].primIdx;

			if (primIdx == -1) {
				t_curr = T_SCENE_MAX;
				break;
			}
			else{
				t_curr = hitArray[i].t;
				int gs_idx = params.gs_idxs[primIdx];

				float o = params.opacity[gs_idx];
				glm::vec3 mean3D = params.means3D[gs_idx];
				glm::mat3x3 SinvR = params.SinvR[gs_idx];

				// Compute intersection point
				glm::vec3 ray_o_mean3D = ray_o - mean3D;
				glm::vec3 o_g = SinvR * ray_o_mean3D; 
				glm::vec3 d_g = SinvR * ray_d;
				float dot_dg_dg = max(1e-6f, glm::dot(d_g, d_g));
				float d = -glm::dot(o_g, d_g) / dot_dg_dg;

				glm::vec3 pos = ray_o + d * ray_d;
				glm::vec3 mean_pos = mean3D - pos;
				glm::vec3 p_g = SinvR * mean_pos; 

				float G = __expf(-0.5f * glm::dot(p_g, p_g));
				float alpha = min(0.99f, o * G);
				if (alpha<params.alpha_min) continue;

				glm::vec3 c = computeColorFromSH_forward(params.deg, ray_d, params.shs + gs_idx * params.max_coeffs);

				float w = T * alpha;
				C += w * c;
				D += w * d;
				O += w;

				T *= (1 - alpha);

				glm::vec3 dL_dc = grad_colors * w;
				float dL_dd = grad_depths * w;
				float dL_dalpha = (
					glm::dot(grad_colors, T * c - (C_final - C)) +
					grad_depths * (T * d - (D_final - D)) + 
					grad_alpha * (1 - O_final)
				) / max(1e-6f, 1 - alpha);
				computeColorFromSH_backward(params.deg, ray_d, params.shs + gs_idx * params.max_coeffs, dL_dc, params.grad_shs + gs_idx * params.max_coeffs);
				float dL_do = dL_dalpha * G;
				float dL_dG = dL_dalpha * o;
				glm::vec3 dL_dpg = -dL_dG * G * p_g;
				glm::mat3x3 dL_dSinvR = glm::outerProduct(dL_dpg, mean_pos);
				// glm::mat3x3 dL_dSinvR = {
				// 	dL_dpg.x * mean_pos.x, dL_dpg.y * mean_pos.x, dL_dpg.z * mean_pos.x, 
				// 	dL_dpg.x * mean_pos.y, dL_dpg.y * mean_pos.y, dL_dpg.z * mean_pos.y, 
				// 	dL_dpg.x * mean_pos.z, dL_dpg.y * mean_pos.z, dL_dpg.z * mean_pos.z
				// };
				
				glm::vec3 dL_dmean_pos = glm::transpose(SinvR) * dL_dpg;
				// vec3 dL_dmean_pos = {
				// 	SinvR[0][0] * dL_dpg.x + SinvR[0][1] * dL_dpg.y + SinvR[0][2] * dL_dpg.z, 
				// 	SinvR[1][0] * dL_dpg.x + SinvR[1][1] * dL_dpg.y + SinvR[1][2] * dL_dpg.z, 
				// 	SinvR[2][0] * dL_dpg.x + SinvR[2][1] * dL_dpg.y + SinvR[2][2] * dL_dpg.z
				// };
				glm::vec3 dL_dmean3D = dL_dmean_pos;

				dL_dd -= glm::dot(dL_dmean_pos, ray_d);

				glm::vec3 dL_dog = -dL_dd / dot_dg_dg * d_g;
				glm::vec3 dL_ddg = -dL_dd / dot_dg_dg * o_g + 2 * dL_dd * glm::dot(o_g, d_g) / max(1e-6f, dot_dg_dg * dot_dg_dg) * d_g;

				dL_dSinvR += glm::outerProduct(dL_dog, ray_o_mean3D);
				// dL_dSinvR += glm::mat3x3{
				// 	dL_dog.x * ray_o_mean3D.x, dL_dog.y * ray_o_mean3D.x, dL_dog.z * ray_o_mean3D.x, 
				// 	dL_dog.x * ray_o_mean3D.y, dL_dog.y * ray_o_mean3D.y, dL_dog.z * ray_o_mean3D.y, 
				// 	dL_dog.x * ray_o_mean3D.z, dL_dog.y * ray_o_mean3D.z, dL_dog.z * ray_o_mean3D.z
				// };

				dL_dmean3D -= glm::transpose(SinvR) * dL_dog;
				//  dL_dmean3D -= vec3{
				// 	SinvR[0][0] * dL_dog.x + SinvR[0][1] * dL_dog.y + SinvR[0][2] * dL_dog.z, 
				// 	SinvR[1][0] * dL_dog.x + SinvR[1][1] * dL_dog.y + SinvR[1][2] * dL_dog.z, 
				// 	SinvR[2][0] * dL_dog.x + SinvR[2][1] * dL_dog.y + SinvR[2][2] * dL_dog.z
				// };

				dL_dSinvR += glm::outerProduct(dL_ddg, ray_d);
				// dL_dSinvR += glm::mat3x3{
				// 	dL_ddg.x * ray_d.x, dL_ddg.y * ray_d.x, dL_ddg.z * ray_d.x, 
				// 	dL_ddg.x * ray_d.y, dL_ddg.y * ray_d.y, dL_ddg.z * ray_d.y, 
				// 	dL_ddg.x * ray_d.z, dL_ddg.y * ray_d.z, dL_ddg.z * ray_d.z
				// };

        		atomic_add((float*)(params.grad_means3D+gs_idx), dL_dmean3D);
				atomicAdd(params.grad_opacity+gs_idx, dL_do);

				float* grad_SinvR = (float*)(params.grad_SinvR + gs_idx);
				for (int j=0; j<9;++j){
					atomicAdd(grad_SinvR+j, dL_dSinvR[j/3][j%3]);
				}

				if (T < params.transmittance_min){
					break;
				}
			}
		}
		if (t_curr==0.0f) break;
		t_start += t_curr;
		if (k>1000){printf("t_curr:%f\n",t_curr);}
	}
}

extern "C" __global__ void __miss__ms() {
}

extern "C" __global__ void __closesthit__ch() {
}

extern "C" __global__ void __anyhit__ah() {
	unsigned int hitArrayPtr0 = optixGetPayload_0();
    unsigned int hitArrayPtr1 = optixGetPayload_1();

    HitInfo* hitArray = (HitInfo*)((uintptr_t)hitArrayPtr0 | ((uintptr_t)hitArrayPtr1 << 32));

	float THit = optixGetRayTmax();
    int i_prim = optixGetPrimitiveIndex();
	HitInfo newHit = {THit, i_prim};

	for (int i = 0; i < MAX_BUFFER_SIZE; ++i) {
        if (hitArray[i].t > newHit.t) {
            // HitInfo temp = hitArray[i];
            // hitArray[i] = newHit;
            // newHit = temp;
			host_device_swap<HitInfo>(hitArray[i], newHit);
        }
    }
	
	// int i = MAX_BUFFER_SIZE - 1;
	// while (i > 0 && hitArray[i - 1].t > newHit.t) {
	// 	hitArray[i] = hitArray[i - 1];
	// 	--i;
	// }
	// hitArray[i] = newHit;


	if (THit < hitArray[MAX_BUFFER_SIZE - 1].t) {
        optixIgnoreIntersection(); 
    }

}

}
